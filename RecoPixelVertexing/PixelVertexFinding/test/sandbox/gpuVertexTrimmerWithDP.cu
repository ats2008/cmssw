#include "hip/hip_runtime.h"
#include "gpuVertexTrimmer.h"
#include "HeterogeneousCore/CUDAUtilities/interface/radixSort.h"


namespace gpuVertexTrimmer {
 
    __global__ void initWorkspaceTr(WorkSpaceTr* pws){
    auto idx =blockIdx.x * blockDim.x + threadIdx.x;
        printf("init idx = %d [MAXVTX = %d ]\n",idx,ZVertexSoA::MAXVTX);
      if (idx < ZVertexSoA::MAXVTX) {
        pws->sumPtt2[idx] = 0.0;
        pws->nTracksFromVertex[idx] = 0;
        pws->newVtxIds[idx] = -1;
      }
    }
      // parallel on all tracks
  __global__ void sumPt2(TkSoA const* ptracks,
                         ZVertexSoA const* pVertexSoa,
                         ZVertexSoA * trimmedVertexSoa,
                         gpuVertexTrimmer::WorkSpaceTr* pws,
                         float ptMin,
                         float ptMax,
                         int minHits) {
    assert(ptracks);
    assert(pVertexSoa);
    assert(pws);

    auto const& tracks = *ptracks;
    auto& vertexs = *pVertexSoa;
    auto& trimmedVertises = *trimmedVertexSoa;
    auto& ws = *pws;
    auto const* quality = tracks.qualityData();

    auto first = blockIdx.x * blockDim.x + threadIdx.x;
 //   printf("@ @sumpT2 with idx  = %d : ZVertexSoA::MAXTRACKS = %d\n ",first, ZVertexSoA::MAXTRACKS);

    //initialization of the track-vertex map for trimmed veretices
    for (int idx = first, nt = TkSoA::stride(); idx < nt; idx += gridDim.x * blockDim.x) 
    {
        trimmedVertises.idv[idx]=-1;
    }
    for (int idx = first, nt = TkSoA::stride(); idx < nt; idx += gridDim.x * blockDim.x) {
      auto nHits = tracks.nHits(idx);
      if (nHits == 0)
        break;  // this is a guard: maybe we need to move to nTracks...

      atomicAdd(&(ws.ntrks), 1);
      auto vid = vertexs.idv[idx];

      auto ndof = 2 * nHits - 5;
      if (ndof < 0)
        ndof = 0;
   //    printf("sumpT2 kernel : checking trk id %d with vid : %d ,chi2 = %f ,ndof = %d, chi2max =%f , pt = %f, ptMin =%f,pTmax = %f \n",
   //            idx,vid,tracks.chi2(idx),ndof,ws.chi2max[ndof],tracks.pt(idx),ptMin,ptMax);
      if (vid < 0)
        continue;

      atomicAdd(&ws.nTracksFromVertex[vid], 1);
      if (nHits < minHits)
        continue;
      if (quality[idx] != trackQuality::loose)
        continue;

      auto chi2 = tracks.chi2(idx);

      if (chi2 > ws.chi2max[ndof])
        continue;

      auto pt = tracks.pt(idx);
      if (pt < ptMin)
        continue;
      if (pt > ptMax)
        pt = ptMax;
      atomicAdd(&ws.sumPtt2[vid], pt * pt);
      //printf("sumpT2 kernel : adding track id %d with vid : %d ,and pT set as : %f [ tempsum =%f ]\n",idx,vid,pt,tempsum);
    }
  }

  // parallel on all vertices
  __global__ void getPt2max(gpuVertexTrimmer::ZVertices const* oVertices,
                            gpuVertexTrimmer::WorkSpaceTr* pws,
                            float fractionSumPt2) {
    auto& vtxSoa = *oVertices;
    auto& ws = *pws;
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto nv = vtxSoa.nvFinal;
    printf("@ getPt2max with idx  = %d : nv = %d\n ",idx,nv);
     
    if (idx == nv - 1) {
        auto sid = vtxSoa.sortInd[idx];
        while (ws.nTracksFromVertex[sid] < 2 and idx > 0) {
          idx--;
          sid = vtxSoa.sortInd[idx];
        }
        ws.maxSumPt2 = ws.sumPtt2[sid] * fractionSumPt2;
      }
    
  }

  // parallel on all vertices
  __global__ void vertexTrimmer(gpuVertexTrimmer::ZVertices* trimmedSoA,
                                gpuVertexTrimmer::ZVertices const* oVertices,
                                gpuVertexTrimmer::WorkSpaceTr* pws,
                                float sumPtMin) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto& origVtxs = *oVertices;
    auto& trimmedVertices = *trimmedSoA;
    auto& ws = *pws;
    auto nv = origVtxs.nvFinal;
    printf("@ vertexTrimmer with idx  = %d : nv = %d\n ",idx,nv);

    if (idx < nv) {
      //        idx=origVtxs.sortInd[idx];
      //printf("ntrks = %d, sumPt2 = %f , sumpTmin = %f , sumPtMax = %f \n",
      //       ws.nTracksFromVertex[idx],
      //       ws.sumPtt2[idx],
      //       sumPtMin,
      //       ws.maxSumPt2);
      if (ws.nTracksFromVertex[idx] > 1 and ws.sumPtt2[idx] > sumPtMin and ws.sumPtt2[idx] > ws.maxSumPt2) {
        auto i = atomicAdd(&trimmedVertices.nvFinal, 1);
        ws.newVtxIds[idx] = i;
        trimmedVertices.zv[i] = origVtxs.zv[idx];
        trimmedVertices.wv[i] = origVtxs.wv[idx];
        trimmedVertices.chi2[i] = origVtxs.chi2[idx];
        trimmedVertices.ptv2[i] = origVtxs.ptv2[idx];
        trimmedVertices.ndof[i] = origVtxs.ndof[idx];
     //   trimmedVertices.sortInd[i] = i;  //TODO
        //printf("adding new i = %d old idx = %d {zv : %f ,wv : %f ,chi2 : %f ,ptv2 : %f ,ndof : %d }\n",
        //       i,
        //       idx,
        //       trimmedVertices.zv[i],
        //       trimmedVertices.wv[i],
        //       trimmedVertices.chi2[i],
        //       trimmedVertices.ptv2[i],
        //       trimmedVertices.ndof[i]);

        //           printf("adding new i = %d old idx = %d ptv2_original =%f , here : %f, max = %f\n",i,idx,trimmedVertices.ptv2[i],ws.sumPtt2[idx],ws.maxSumPt2);
      }
      //    else
      //    {
      //        ws.newVtxIds[idx]=-1;
      //    }
    }
  }

  // parallel on tracks
  __global__ void updateTrackVertexMap(gpuVertexTrimmer::ZVertices const* oVertices,
                                       gpuVertexTrimmer::ZVertices* tVertices,
                                       gpuVertexTrimmer::WorkSpaceTr* pws) {
    assert(oVertices);
    assert(tVertices);
    assert(pws);

    auto& pVertices = *oVertices;
    auto& trimmedVertices = *tVertices;
    auto& ws = *pws;

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("@ updateTrackVertexMap with idx  = %d : ws.ntrks = %d\n ",idx,ws.ntrks);
    if (idx < ws.ntrks) {
      auto vid = pVertices.idv[idx];
      if (vid > -1)
        trimmedVertices.idv[idx] = ws.newVtxIds[vid];
      // printf("updating vertex map of %d[%d], from %d to %d \n ",idx,ws.ntrks,vid,trimmedVertices.idv[idx]) ;
    }
  }

__global__ void sortByPt2(ZVertices* pdata) {

    auto& __restrict__ data = *pdata;
    uint32_t const& nvFinal = data.nvFinal;
    float* __restrict__ ptv2 = data.ptv2;
    uint16_t* __restrict__ sortInd = data.sortInd;

    if (nvFinal < 1)
      return;

    if (1 == nvFinal) {
      if (threadIdx.x == 0)
        sortInd[0] = 0;
      return;
    }
    __shared__ uint16_t sws[1024];
    
    // sort using only 16 bits
    radixSort<float, 2>(ptv2, sortInd, sws, nvFinal);
    
    }

  __global__ void printVtx(gpuVertexTrimmer::ZVertices* tVertices)
  {
    assert(tVertices);

    auto& trimmedVertices = *tVertices;

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < trimmedVertices.nvFinal) {
      printf("idx = %d ptt2 = %f , sortInd = %d  \n ",idx,trimmedVertices.ptv2[idx],trimmedVertices.sortInd[idx]) ;
  }
  }


    __global__ void initWorkspaceTrWrapper(WorkSpaceTr* pws,const ZVertexSoA* Verteices  ){
        auto nv= Verteices->nvFinal;
        auto blockSize=128;
        auto numberOfBlocks =(nv +blockSize -1)/blockSize;
        if(nv<blockSize) blockSize=nv;
        initWorkspaceTr<<<numberOfBlocks,blockSize,0>>>(pws);
    }

   __global__ void getPt2maxWraper(gpuVertexTrimmer::ZVertices const* oVertices,
                            gpuVertexTrimmer::WorkSpaceTr* pws,
                            float fractionSumPt2) {
        auto nv= oVertices->nvFinal;
        auto blockSize=128;
        auto numberOfBlocks =(nv +blockSize -1)/blockSize;
        if(nv<blockSize) blockSize=nv;
       getPt2maxWraper<<<numberOfBlocks,blockSize,0>>>(oVertices,pws,fractionSumPt2);
  }

  __global__ void vertexTrimmerWraper(gpuVertexTrimmer::ZVertices* trimmedSoA,
                                gpuVertexTrimmer::ZVertices const* oVertices,
                                gpuVertexTrimmer::WorkSpaceTr* pws,
                                float sumPtMin) {
        auto nv= oVertices->nvFinal;
        auto blockSize=128;
        auto numberOfBlocks =(nv +blockSize -1)/blockSize;
        if(nv<blockSize) blockSize=nv;
       vertexTrimmer<<<numberOfBlocks,blockSize,0>>>(trimmedSoA, oVertices, pws,sumPtMin);
  }

  __global__ void updateTrackVertexMapWraper(gpuVertexTrimmer::ZVertices const* oVertices,
                                       gpuVertexTrimmer::ZVertices* tVertices,
                                       gpuVertexTrimmer::WorkSpaceTr* pws) {
        auto nt= pws->ntrks;
        auto blockSize=128;
        auto numberOfBlocks =(nt +blockSize -1)/blockSize;
        if(nt<blockSize) blockSize=nt;
        updateTrackVertexMap<<<numberOfBlocks,blockSize,0>>>(oVertices,tVertices,pws);

  }

  void setMaxChi2(size_t ndof, float* maxChi2, float track_prob_min, float maxChi2set);

  ZVertexHeterogeneous Trimmer::makeAsync(hipStream_t stream, TkSoA const* tksoa, ZVertexSoA const* VertexSoA) const {
    std::cout << " Starting async func \n ";
    ZVertexHeterogeneous vertices(cms::cuda::make_device_unique<ZVertexSoA>(stream));

    auto* trimmedVertexSoA = vertices.get();
    //std::cout<<"Going for asserts : ";
    assert(VertexSoA);
    //std::cout<<" VertexSoA done ";
    assert(tksoa);
    //std::cout<<" tksoa done ";
    assert(trimmedVertexSoA);
    std::cout << " trimmedVertexSoA done \n";

    auto ws_tr = cms::cuda::make_device_unique<WorkSpaceTr>(stream);
    auto* workspace = ws_tr.get();

    auto maxchi2valsCPU = cms::cuda::make_host_unique<float[]>(20, stream);
    setMaxChi2(20, maxchi2valsCPU.get(), track_prob_min_, chi2Max_);
    std::cout<<"Chi2 Max Values made as \n" ;
    for(int i=0;i<8;i++)
        std::cout<<"ndof  = "<<i<<" chi2Max = "<<maxchi2valsCPU[i]<<"\n";
    hipMemcpyAsync(workspace->chi2max,
                    maxchi2valsCPU.get(),
                    20 * sizeof(float),
                    hipMemcpyHostToDevice,
                    stream);

    init<<<1, 1, 0, stream>>>(trimmedVertexSoA, workspace);
    initWorkspaceTrWrapper<<<1,1,0,stream>>>(workspace,VertexSoA);
    cudaCheck(hipGetLastError());

    auto blockSize = 128;

    auto numberOfBlocks = (TkSoA::stride() + blockSize - 1) / blockSize;

    printf("@ sumpT2 launched with numberOfBlocks  = %d : blockSize = %d\n ",numberOfBlocks,blockSize );
    sumPt2<<<numberOfBlocks, blockSize, 0, stream>>>(tksoa, VertexSoA, trimmedVertexSoA, workspace, track_pT_min_, track_pT_max_, minNumberOfHits_);
    cudaCheck(hipGetLastError());
    
    printf("@getPt2maxWraper launched\n " );
    getPt2maxWraper<<<1,1,0, stream>>>(VertexSoA, workspace, fractionSumPt2_);
    cudaCheck(hipGetLastError());
    printf("@getPt2maxWraper sucessfully done\n " );

    printf("@vertexTrimmerWraper launched\n " );
    vertexTrimmerWraper<<<1,1,0, stream>>>(trimmedVertexSoA, VertexSoA, workspace, minSumPt2_);
    cudaCheck(hipGetLastError());
    printf("@vertexTrimmerWraper done\n " );
   
    printf("@updateTrackVertexMapWraper launched\n " );
    updateTrackVertexMapWraper<<<1,1,0, stream>>>(VertexSoA, trimmedVertexSoA, workspace);
    cudaCheck(hipGetLastError());
    printf("@updateTrackVertexMapWraper done\n " );
    
    printf("@sortByPt2 launched\n " );
    sortByPt2<<<1,1024 - 256,0,stream>>>(trimmedVertexSoA);
    cudaCheck(hipGetLastError());
    printf("@sortByPt2 done\n " );
    
    printf("@peintVtx launched\n " );
    printVtx<<<numberOfBlocks,blockSize,0,stream>>>(trimmedVertexSoA);
    cudaCheck(hipGetLastError());
    
    std::cout << "exiting AsyncFuc \n";

    return vertices;
  }
 
  void setMaxChi2(size_t ndof, float* maxChi2, float track_prob_min, float maxChi2set) {
    if (track_prob_min > 0 and track_prob_min <= 1.0) {
      for (size_t i = 1; i <= ndof; i++) {
        maxChi2[i] = TMath::ChisquareQuantile(1 - track_prob_min, i) / i;
        if (maxChi2[i] > maxChi2set)
          maxChi2[i] = maxChi2set;
      }
    } else {
      for (size_t i = 0; i <= ndof; i++) {
        maxChi2[i] = maxChi2set;
      }
    }
    maxChi2[0] = -1.0;
  }


}  // namespace gpuVertexTrimmer

