#include "hip/hip_runtime.h"
#include "gpuVertexTrimmer.h"
#include "HeterogeneousCore/CUDAUtilities/interface/radixSort.h"


namespace gpuVertexTrimmer {
 
    __global__ void initWorkspaceTr(WorkSpaceTr* pws){
    auto idx =blockIdx.x * blockDim.x + threadIdx.x;
        printf("init idx = %d [MAXVTX = %d ]\n",idx,ZVertexSoA::MAXVTX);
      if (idx < ZVertexSoA::MAXVTX) {
        pws->sumPtt2[idx] = 0.0;
        pws->nTracksFromVertex[idx] = 0;
      }
    }
    __global__ void initWorkspaceTrWrapper(WorkSpaceTr* pws,const ZVertexSoA* Verteices  ){

        auto nv= Verteices->nvFinal;
        auto blockSize=128;
        auto numberOfBlocks =(nv +blockSize -1)/blockSize;
        if(nv<blockSize) blockSize=nv;
        initWorkspaceTr<<<numberOfBlocks,blockSize,0>>>(pws);
    }

    __global__ void sortByPt2(ZVertices* pdata) {

    auto& __restrict__ data = *pdata;
    uint32_t const& nvFinal = data.nvFinal;
    float* __restrict__ ptv2 = data.ptv2;
    uint16_t* __restrict__ sortInd = data.sortInd;

    if (nvFinal < 1)
      return;

    if (1 == nvFinal) {
      if (threadIdx.x == 0)
        sortInd[0] = 0;
      return;
    }
    __shared__ uint16_t sws[1024];
    
    // sort using only 16 bits
    radixSort<float, 2>(ptv2, sortInd, sws, nvFinal);
    
    }

  
  // parallel on all tracks
  __global__ void sumPt2(TkSoA const* ptracks,
                         ZVertexSoA const* pVertexSoa,
                         gpuVertexTrimmer::WorkSpaceTr* pws,
                         float ptMin,
                         float ptMax,
                         int minHits) {
    assert(ptracks);
    assert(pVertexSoa);
    assert(pws);

    auto const& tracks = *ptracks;
    auto& vertexs = *pVertexSoa;
    auto& ws = *pws;
    auto const* quality = tracks.qualityData();

    auto first = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = first, nt = TkSoA::stride(); idx < nt; idx += gridDim.x * blockDim.x) {
      auto nHits = tracks.nHits(idx);
      if (nHits == 0)
        break;  // this is a guard: maybe we need to move to nTracks...

      atomicAdd(&(ws.ntrks), 1);
      auto vid = vertexs.idv[idx];

      auto ndof = 2 * nHits - 5;
      if (ndof < 0)
        ndof = 0;
   //    printf("sumpT2 kernel : checking trk id %d with vid : %d ,chi2 = %f ,ndof = %d, chi2max =%f , pt = %f, ptMin =%f,pTmax = %f \n",
   //            idx,vid,tracks.chi2(idx),ndof,ws.chi2max[ndof],tracks.pt(idx),ptMin,ptMax);
      if (vid < 0)
        continue;

      atomicAdd(&ws.nTracksFromVertex[vid], 1);
      if (nHits < minHits)
        continue;
      if (quality[idx] != trackQuality::loose)
        continue;

      auto chi2 = tracks.chi2(idx);

      if (chi2 > ws.chi2max[ndof])
        continue;

      auto pt = tracks.pt(idx);
      if (pt < ptMin)
        continue;
      if (pt > ptMax)
        pt = ptMax;
      atomicAdd(&ws.sumPtt2[vid], pt * pt);
      //printf("sumpT2 kernel : adding track id %d with vid : %d ,and pT set as : %f [ tempsum =%f ]\n",idx,vid,pt,tempsum);
    }
  }

  // parallel on all vertices
  __global__ void getPt2max(gpuVertexTrimmer::ZVertices const* oVertices,
                            gpuVertexTrimmer::WorkSpaceTr* pws,
                            float fractionSumPt2) {
    auto& vtxSoa = *oVertices;
    auto& ws = *pws;
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto nv = vtxSoa.nvFinal;

    if (idx < nv) {
      ws.newVtxIds[idx] = -1;
      //     idx=vtxSoa.sortInd[idx];
     // printf("in getPt2max , idx = %d , sortIdx = %d ->nv = %d  sumpT2 =%f \n",
     //        idx,
     //        vtxSoa.sortInd[idx],
     //        nv,
     //        ws.sumPtt2[idx]);
      if (idx == nv - 1) {
        auto sid = vtxSoa.sortInd[idx];
        while (ws.nTracksFromVertex[sid] < 2 and idx > 0) {
          idx--;
          sid = vtxSoa.sortInd[idx];
        }
        //           idx=vtxSoa.sortInd[idx];
        ws.maxSumPt2 = ws.sumPtt2[sid] * fractionSumPt2;
       // printf(
       //     "in getPt2max  FOUND MAX AS : @idx = %d[%d] ,pTMax = %f [ %f ]\n ", idx, nv, ws.maxSumPt2, ws.sumPtt2[sid]);
        //printf("found sortId = %d , ws.maxSumPt2=ws.sumPtt2[%d]*fractionSumPt2 :: %f=%f * %f \n",
        //                      idx,idx,ws.maxSumPt2,ws.sumPtt2[idx],fractionSumPt2);
      }
    }
  }

  // parallel on all vertices
  __global__ void vertexTrimmer(gpuVertexTrimmer::ZVertices* trimmedSoA,
                                gpuVertexTrimmer::ZVertices const* oVertices,
                                gpuVertexTrimmer::WorkSpaceTr* pws,
                                float sumPtMin) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto& origVtxs = *oVertices;
    auto& trimmedVertices = *trimmedSoA;
    auto& ws = *pws;
    auto nv = origVtxs.nvFinal;

    if (idx < nv) {
      //        idx=origVtxs.sortInd[idx];
      //printf("ntrks = %d, sumPt2 = %f , sumpTmin = %f , sumPtMax = %f \n",
      //       ws.nTracksFromVertex[idx],
      //       ws.sumPtt2[idx],
      //       sumPtMin,
      //       ws.maxSumPt2);
      if (ws.nTracksFromVertex[idx] > 1 and ws.sumPtt2[idx] > sumPtMin and ws.sumPtt2[idx] > ws.maxSumPt2) {
        auto i = atomicAdd(&trimmedVertices.nvFinal, 1);
        ws.newVtxIds[idx] = i;
        trimmedVertices.zv[i] = origVtxs.zv[idx];
        trimmedVertices.wv[i] = origVtxs.wv[idx];
        trimmedVertices.chi2[i] = origVtxs.chi2[idx];
        trimmedVertices.ptv2[i] = origVtxs.ptv2[idx];
        trimmedVertices.ndof[i] = origVtxs.ndof[idx];
     //   trimmedVertices.sortInd[i] = i;  //TODO
        //printf("adding new i = %d old idx = %d {zv : %f ,wv : %f ,chi2 : %f ,ptv2 : %f ,ndof : %d }\n",
        //       i,
        //       idx,
        //       trimmedVertices.zv[i],
        //       trimmedVertices.wv[i],
        //       trimmedVertices.chi2[i],
        //       trimmedVertices.ptv2[i],
        //       trimmedVertices.ndof[i]);

        //           printf("adding new i = %d old idx = %d ptv2_original =%f , here : %f, max = %f\n",i,idx,trimmedVertices.ptv2[i],ws.sumPtt2[idx],ws.maxSumPt2);
      }
      //    else
      //    {
      //        ws.newVtxIds[idx]=-1;
      //    }
    }
  }

  // parallel on all tracks
  __global__ void updateTrackVertexMap(gpuVertexTrimmer::ZVertices const* oVertices,
                                       gpuVertexTrimmer::ZVertices* tVertices,
                                       gpuVertexTrimmer::WorkSpaceTr* pws) {
    assert(oVertices);
    assert(tVertices);
    assert(pws);

    auto& pVertices = *oVertices;
    auto& trimmedVertices = *tVertices;
    auto& ws = *pws;

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < ZVertexSoA::MAXVTX)
      trimmedVertices.idv[idx] = -1;

    if (idx < ws.ntrks) {
      auto vid = pVertices.idv[idx];
      if (vid > -1)
        trimmedVertices.idv[idx] = ws.newVtxIds[vid];

      // printf("updating vertex map of %d[%d], from %d to %d \n ",idx,ws.ntrks,vid,trimmedVertices.idv[idx]) ;
    }
  }

  __global__ void printVtx(gpuVertexTrimmer::ZVertices* tVertices)
  {
    assert(tVertices);

    auto& trimmedVertices = *tVertices;

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < trimmedVertices.nvFinal) {
      printf("idx = %d ptt2 = %f , sortInd = %d  \n ",idx,trimmedVertices.ptv2[idx],trimmedVertices.sortInd[idx]) ;
  }
  }


  void updateMaxChi2(size_t ndof, float* maxChi2, float track_prob_min, float maxChi2set) {
    if (track_prob_min > 0 and track_prob_min <= 1.0) {
      for (size_t i = 1; i <= ndof; i++) {
        maxChi2[i] = TMath::ChisquareQuantile(1 - track_prob_min, i) / i;
        if (maxChi2[i] > maxChi2set)
          maxChi2[i] = maxChi2set;
      }
    } else {
      for (size_t i = 0; i <= ndof; i++) {
        maxChi2[i] = maxChi2set;
      }
    }
    maxChi2[0] = -1.0;

    //  for (size_t i =0; i <=ndof; i++)
    //  std::cout<<" ndof = "<<i<<maxChi2[i]<<"\n";
  }

  ZVertexHeterogeneous Trimmer::makeAsync(hipStream_t stream, TkSoA const* tksoa, ZVertexSoA const* VertexSoA) const {
    std::cout << " Starting async func \n ";
    ZVertexHeterogeneous vertices(cms::cuda::make_device_unique<ZVertexSoA>(stream));

    auto* trimmedVertexSoA = vertices.get();
    //std::cout<<"Going for asserts : ";
    assert(VertexSoA);
    //std::cout<<" VertexSoA done ";
    assert(tksoa);
    //std::cout<<" tksoa done ";
    assert(trimmedVertexSoA);
    std::cout << " trimmedVertexSoA done \n";

    auto ws_tr = cms::cuda::make_device_unique<WorkSpaceTr>(stream);
    auto* workspace = ws_tr.get();

    auto maxchi2valsCPU = cms::cuda::make_host_unique<float[]>(20, stream);
    updateMaxChi2(20, maxchi2valsCPU.get(), track_prob_min_, chi2Max_);
    std::cout<<"Chi2 Max Values made as \n" ;
    for(int i=0;i<8;i++)
        std::cout<<"ndof  = "<<i<<" chi2Max = "<<maxchi2valsCPU[i]<<"\n";
    hipMemcpyAsync(workspace->chi2max,
                    maxchi2valsCPU.get(),
                    20 * sizeof(float),
                    hipMemcpyHostToDevice,
                    stream);

    //std::cout<<"Going to the init kernel \n"; //with numberOfBlocks = "<<numberOfBlocks<<" mN"<<minNumberOfHits_<<" c2M "<<chi2Max_<<"\n";
    init<<<1, 1, 0, stream>>>(trimmedVertexSoA, ws_tr.get());

    initWorkspaceTrWrapper<<<1,1,0,stream>>>(workspace,VertexSoA);
    
    //std::cout<<"out of init kernel \n";
    cudaCheck(hipGetLastError());
    std::cout << "passed checkerror of init kernel \n";

    auto blockSize = 128;

    auto numberOfBlocks = (TkSoA::stride() + blockSize - 1) / blockSize;
    //   loadTracksForTrimmer<<<numberOfBlocks, blockSize, 0, stream>>>(tksoa, trimmedVertexSoA, workspace, track_pT_min_);
    //   cudaCheck(hipGetLastError());

    //std::cout<<"Going to the sumPt2 kernel with numberOfBlocks = "<<numberOfBlocks<<" mN"<<minNumberOfHits_<<" c2M "<<chi2Max_<<"\n";
    sumPt2<<<numberOfBlocks, blockSize, 0, stream>>>(
        tksoa, VertexSoA, workspace, track_pT_min_, track_pT_max_, minNumberOfHits_);
    //std::cout<<"out of sumPt2 kernel \n";
    cudaCheck(hipGetLastError());
    // std::cout<<"passed checkerror of sumPt2 kernel \n";

    //std::cout<<"Going to the getPt2max kernel with numberOfBlocks = "<<numberOfBlocks<<"  fractionSumPt2 = "<<fractionSumPt2_<<"\n";
    getPt2max<<<numberOfBlocks, blockSize, 0, stream>>>(VertexSoA, workspace, fractionSumPt2_);
    // std::cout<<"out of getPt2Max kernel \n";
    cudaCheck(hipGetLastError());
    //std::cout<<"passed checkerror of getPt2max kernel \n";

    //std::cout<<"updating the number of Blocks now with VertexSoA->nvFinal - ...\n ";
    numberOfBlocks = (ZVertexSoA::MAXVTX + blockSize - 1) / blockSize;
    //std::cout<<"Going to the vertextrimmer kernel with numberOfBlocks = "<<numberOfBlocks<<"\n";
    vertexTrimmer<<<numberOfBlocks, blockSize, 0, stream>>>(trimmedVertexSoA, VertexSoA, workspace, minSumPt2_);
    //std::cout<<"out of vertexTrimmer kernel \n";
    cudaCheck(hipGetLastError());
    //std::cout<<"passed checkerror of vertextrimmer kernel \n";

    numberOfBlocks = (TkSoA::stride() + blockSize - 1) / blockSize;
    //std::cout<<"Going to the updateTrackVertexMap kernel with numberOfBlocks = "<<numberOfBlocks<<"\n";
    updateTrackVertexMap<<<numberOfBlocks, blockSize, 0, stream>>>(VertexSoA, trimmedVertexSoA, workspace);
    //std::cout<<"out of updateTrackVertexMap kernel \n";
    cudaCheck(hipGetLastError());
    //std::cout<<"passed checkerror of updateTrackVertexMap kernel \n";
    numberOfBlocks = (ZVertexSoA::MAXVTX + blockSize - 1) / blockSize;
    std::cout<<"going to sort by pT2 \n";
    sortByPt2<<<1,1024 - 256,0,stream>>>(trimmedVertexSoA);
    std::cout<<"passed checkerror of sumpt2 kernel \n";
    std::cout<<"going to printVtx \n";
    printVtx<<<numberOfBlocks,blockSize,0,stream>>>(trimmedVertexSoA);
    std::cout<<"passed checkerror of printVtx kernel \n";
    cudaCheck(hipGetLastError());
    std::cout << "exiting AsyncFuc \n";

    return vertices;
  }

}  // namespace gpuVertexTrimmer

